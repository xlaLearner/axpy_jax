#include "hip/hip_runtime.h"
/*
这里是对axpy的GPU实现
*/

#include "axpy.h"
#include "axpy_gpu_kernel.h"
#include "axpy_gpu_kernel_helpers.h"

namespace axpy_jax {

namespace {

template <template T>
__global__ void axpy_kernel(const std::int64_t a, const T *x, const T *y, const T *result) {
    for (std::int64_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < size;
       idx += blockDim.x * gridDim.x) {
        compute_axpy<T>(a, x, y, result);
    }
}

void ThrowIfError(hipError_t error) {
  if (error != hipSuccess) {
    throw std::runtime_error(hipGetErrorString(error));
  }
}

template <typename T>
inline void apply_kepler(hipStream_t stream, void **buffers, const char *opaque,
                         std::size_t opaque_len) {
  const KeplerDescriptor &d = *UnpackDescriptor<AxpyDescriptor>(opaque, opaque_len);
  const std::int64_t size = d.size;

  const std::int64_t a = *reinterpret_cast<const T>(buffers[0]);
  const T *x = reinterpret_cast<const T *>(buffers[1]);
  const T *y = reinterpret_cast<const T *>(buffers[2]);

  const T *result = reinterpret_cast<T *>(buffers[3]);

  // 可以考虑如何设置这些使得分配的软件更适合算法
  const int block_dim = 128;
  const int grid_dim = std::min<int>(1024, (size + block_dim - 1) / block_dim);
  axpy_kernel<T>
      <<<grid_dim, block_dim, 0, stream>>>(a, x, y, result);

  ThrowIfError(hipGetLastError());
}

}

void gpu_axpy_f32(hipStream_t stream, void **buffers, const char *opaque,
                    std::size_t opaque_len) {
  apply_axpy<float>(stream, buffers, opaque, opaque_len);
}

void gpu_axpy_f64(hipStream_t stream, void **buffers, const char *opaque,
                    std::size_t opaque_len) {
  apply_axpy<double>(stream, buffers, opaque, opaque_len);
}

}